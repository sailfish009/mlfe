#include "hip/hip_runtime.h"
#include "functions_cuda.hpp"
#include "functions.hpp"
#include <cub\block\block_reduce.cuh>
#include "../device_context/cuda_context.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
namespace mlfe { namespace math {

template <typename T>
__global__ void random_uniform_shift_kernel(
    const unsigned int size,
    T *numbers, const T a, const T b)
{
    const T scale = b - a;
    CUDA_1D_KERNEL_LOOP(n, size) {
        numbers[n] = numbers[n] * scale + a;
    }
}

template <>
void UniformCurand<float>(
    hiprandGenerator_t *gen,
    const unsigned int size,
    float *numbers, const float a, const float b)
{
    hiprandGenerateUniform(*gen, numbers, size);
    random_uniform_shift_kernel<float><<<
        CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, numbers, a, b);
}

template <class DataType>
__global__ void ReLUKernel(const int size, const DataType *x, DataType *y) {
    CUDA_1D_KERNEL_LOOP(i, size) {
        y[i] = x[i] > 0 ? x[i] : 0;
    }
}

template <class DataType>
__global__ void ReLUGradientKernel(const int size, const DataType *x, const DataType *dy, DataType *dx) {
    CUDA_1D_KERNEL_LOOP(i, size) {
        dx[i] = x[i] > 0 ? dy[i] : 0;
    }
}

template <>
void ReluFunction<float, HIPContext>(const int size, const float *x, float *y) {
    ReLUKernel<float> << <CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS >> >(size, x, y);
}

template <>
void ReluGradientFunction<float, HIPContext>(const int size, const float *x, const float *dy, float *dx) {
    ReLUGradientKernel<float><<<CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, x, dy, dx);
}


template <class DataType>
__global__ void SigmoidKernel(const int size, const DataType *x, DataType *y) {
    CUDA_1D_KERNEL_LOOP(i, size) {
        y[i] = 1.f / (1.f + exp(-x[i]));
    }
}

template <class DataType>
__global__  void SigmoidGradientKernel(const int size, const DataType *y, const DataType *dy, DataType *dx) {
    CUDA_1D_KERNEL_LOOP(i, size) {
        dx[i] = dy[i] * y[i] * (1. - y[i]);
    }
}

template <>
void SigmoidFunction<float, HIPContext>(const int size, const float *x, float *y) {
    SigmoidKernel<float><<<
        CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, x, y);
}

template <>
void SigmoidGradientFunction<float, HIPContext>(const int size, const float *y, const float *dy, float *dx) {
    SigmoidGradientKernel<float><<<
        CUDA_CONTEXT_GET_BLOCKS(size),
        CUDA_CONTEXT_NUM_THREADS>>>(size, y, dy, dx);
}

template <typename T>
__global__ void one_hot_kernel(const int classes, const T *label, T *onehot) {
    int n = threadIdx.x;
    int label_val = static_cast<int>(label[n]);
    onehot[n * classes + label_val] = static_cast<T>(1);
}

template <>
void OneHotCuda<float>(const int batch, const int classes, const float *label, float *onehot) {
    one_hot_kernel<float><<<1, batch>>>(classes, label, onehot);
}


template <typename T>
__global__ void divide_by_val_kernel(const int val, T *arg) {
    arg[0] = arg[0] / static_cast<T>(val);
}

template <typename T>
__global__ void top_k_correct_count_kernel(const int batch, const int classes, const int top_k, const T *prob, const T *label, T *accuracy) {
    typedef hipcub::BlockReduce<int, CUDA_CONTEXT_NUM_THREADS> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    int correct = 0;
    for (int b = blockIdx.x; b < batch; b += gridDim.x) {
        const int gt = static_cast<int>(label[b]);
        const T gt_prob = prob[b * classes + gt];
        int rank = 0;
        for (int n = threadIdx.x; n < classes; n += blockDim.x) {
            const T prob_ = prob[b * classes + n];
            if (prob_ > gt_prob) {
                ++rank;
            }
        }
        rank = BlockReduce(temp_storage).Sum(rank);
        if (rank < top_k) {
            ++correct;
        }
        __syncthreads();
    }

    if (threadIdx.x == 0) {
        atomicAdd(accuracy, static_cast<T>(correct));
    }
}

template <>
void AccuracyCuda<float>(const int batch, const int classes, const int top_k, const float *prob, const float *label, float *accuracy) {
    top_k_correct_count_kernel<float><<<
        CUDA_CONTEXT_GET_BLOCKS(batch * classes),
        CUDA_CONTEXT_NUM_THREADS>>>(
            batch, classes, top_k, prob, label, accuracy
            );
    divide_by_val_kernel<float><<<1, 1>>>(batch, accuracy);
}

} // end namespace math
} // end namespace mlfe
