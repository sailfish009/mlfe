#include "hip/hip_runtime.h"
#include "functions_cuda.hpp"
#include <cub\block\block_reduce.cuh>
#include "../device_context/cuda_context.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
namespace mlfe { namespace math {

__global__ void curand_init_kernel(const int size, unsigned int seed, hiprandState_t *states) {
    CUDA_1D_KERNEL_LOOP(n, size) {
        hiprand_init(seed, n, 0, &states[n]);
    }
}

template <typename T>
__global__ void curand_uniform_kernel(hiprandState_t *states, const int size, T *numbers, T a, T b) {
    CUDA_1D_KERNEL_LOOP(n, size) {
        numbers[n] = hiprand_uniform(&states[n]);
        numbers[n] = numbers[n] * (b - a) + a;
    }
}

void InitCurand(unsigned int seed, unsigned int n, hiprandState_t* states) {
    curand_init_kernel<<<CUDA_CONTEXT_GET_BLOCKS(n),
        CUDA_CONTEXT_NUM_THREADS >>>(n, seed, states);
}

template <>
void UniformCurand<float>(hiprandState_t *states, unsigned int n, float *numbers, float a, float b) {
    curand_uniform_kernel<float><<<CUDA_CONTEXT_GET_BLOCKS(n),
        CUDA_CONTEXT_NUM_THREADS >>>(states, n, numbers, a, b);
}


} // end namespace math
} // end namespace mlfe
